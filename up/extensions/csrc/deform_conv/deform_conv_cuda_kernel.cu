#include "hip/hip_runtime.h"
#include "deform_conv/deformable_conv.h"

using at::Tensor;
using at::Half;

#define CUDA_KERNEL_LOOP(i, n)                                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);                 \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;

inline int GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

#ifndef CAFFE_COMMON_CUH_
#define CAFFE_COMMON_CUH_

#include <hip/hip_runtime.h>

  #if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

  #else//performence loss
      static __inline__ __device__ double atomicAdd(double *address, double val) {
        unsigned long long int* address_as_ull = (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;
        if (val==0.0)
          return __longlong_as_double(old);
        do {
          assumed = old;
          old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
        } while (assumed != old);
        return __longlong_as_double(old);
      }
  #endif
#endif

static __inline__ __device__ at::Half atomicAdd(at::Half* address, at::Half val) {
  unsigned int *aligned = (unsigned int*)((size_t)address - ((size_t)address & 2));
  unsigned int old = *aligned;
  unsigned int assumed;
  unsigned short old_as_us;
  do {
    assumed = old;
    old_as_us = (unsigned short)((size_t)address & 2 ? old >> 16 : old & 0xffff);
#if __CUDACC_VER_MAJOR__ >= 9
    half sum = __float2half_rn(__half2float(__ushort_as_half(old_as_us)) + float(val));
    unsigned short sum_as_us = __half_as_ushort(sum);
#else
    unsigned short sum_as_us = __float2half_rn(__half2float(old_as_us) + float(val));
#endif
    unsigned int sum_as_ui = (size_t)address & 2 ? (sum_as_us << 16) | (old & 0xffff)
                                                 : (old & 0xffff0000) | sum_as_us;
    old = atomicCAS(aligned, assumed, sum_as_ui);
  } while(assumed != old);
  //__half_raw raw = {old_as_us};
  //return at::Half(raw);
  return at::Half({__ushort_as_half(old_as_us)});
};


template <typename scalar_t>
__device__ scalar_t deformable_im2col_bilinear(const scalar_t *bottom_data,
                                               const int data_width,
                                               const int height, const int width,
                                               scalar_t h, scalar_t w) {

  int h_low = floor(h);
  int w_low = floor(w);
  int h_high;
  int w_high;
  if (h_low >= height - 1) {
    h_high = h_low = height - 1;
    h = (scalar_t)h_low;
  } else {
    h_high = h_low + 1;
  }

  if (w_low >= width - 1) {
    w_high = w_low = width - 1;
    w = (scalar_t)w_low;
  } else {
    w_high = w_low + 1;
  }

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = bottom_data[h_high * data_width + w_high];
  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(scalar_t argmax_h, scalar_t argmax_w,
                                     const int h, const int w, const int height,
                                     const int width) {

  if (argmax_h < 0 || argmax_h > height || argmax_w < 0 || argmax_w > width) {
    // empty
    return 0;
  }

  argmax_h = max(argmax_h, (scalar_t)0.0f);
  argmax_w = max(argmax_w, (scalar_t)0.0f);

  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (scalar_t)argmax_h_low;
  } else {
    argmax_h_high = argmax_h_low + 1;
  }
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (scalar_t)argmax_w_low;
  } else {
    argmax_w_high = argmax_w_low + 1;
  }
  scalar_t weight = 0;
  if (h == argmax_h_low) {
    if (w == argmax_w_low) {
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
    } else if (w == argmax_w_high) {
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
    }
  } else if (h == argmax_h_high) {
    if (w == argmax_w_low) {
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
    } else if (w == argmax_w_high) {
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
    }
  }
  return weight;
}

template <typename scalar_t>
__device__ scalar_t get_coordinate_weight(scalar_t argmax_h, scalar_t argmax_w,
                                          const int height, const int width,
                                          const scalar_t *im_data,
                                          const int data_width, const int bp_dir) {

  if (argmax_h < 0 || argmax_h > height || argmax_w < 0 || argmax_w > width) {
    // empty
    return 0;
  }

  if (argmax_h < 0)
    argmax_h = 0;
  if (argmax_w < 0)
    argmax_w = 0;

  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (scalar_t)argmax_h_low;
  } else {
    argmax_h_high = argmax_h_low + 1;
  }
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (scalar_t)argmax_w_low;
  } else {
    argmax_w_high = argmax_w_low + 1;
  }
  scalar_t weight = 0;

  if (bp_dir == 0) {
    weight += -1 * (argmax_w_low + 1 - argmax_w) *
              im_data[argmax_h_low * data_width + argmax_w_low];
    weight += -1 * (argmax_w - argmax_w_low) *
              im_data[argmax_h_low * data_width + argmax_w_high];
    weight += (argmax_w_low + 1 - argmax_w) *
              im_data[argmax_h_high * data_width + argmax_w_low];
    weight += (argmax_w - argmax_w_low) *
              im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    weight += -1 * (argmax_h_low + 1 - argmax_h) *
              im_data[argmax_h_low * data_width + argmax_w_low];
    weight += (argmax_h_low + 1 - argmax_h) *
              im_data[argmax_h_low * data_width + argmax_w_high];
    weight += -1 * (argmax_h - argmax_h_low) *
              im_data[argmax_h_high * data_width + argmax_w_low];
    weight += (argmax_h - argmax_h_low) *
              im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename scalar_t>
__global__ void deformable_im2col_gpu_kernel(
    const int n, const scalar_t *data_im, const scalar_t *data_offset,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int height_col,
    const int width_col, scalar_t *data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int c_im = (index / width_col) / height_col;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;
    scalar_t *data_col_ptr =
        data_col + (c_col * height_col + h_col) * width_col + w_col;
    const scalar_t *data_im_ptr = data_im + (c_im * height + h_in) * width + w_in;
    const scalar_t *data_offset_ptr = data_offset + deformable_group_index * 2 *
                                                     kernel_h * kernel_w *
                                                     height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
        const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
        scalar_t val = static_cast<scalar_t>(0);
        const scalar_t h_im = h_in + i * dilation_h + offset_h;
        const scalar_t w_im = w_in + j * dilation_w + offset_w;
        if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
          const scalar_t map_h = i * dilation_h + offset_h;
          const scalar_t map_w = j * dilation_w + offset_w;
          const int cur_height = height - h_in;
          const int cur_width = width - w_in;
          val = deformable_im2col_bilinear<scalar_t>(data_im_ptr, width, cur_height,
                                           cur_width, map_h, map_w);
        }
        *data_col_ptr = val;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

void deformable_im2col(Tensor data_im,
                       Tensor data_offset, const int channels,
                       const int height, const int width, const int ksize_h, const int ksize_w, const int pad_h, const int pad_w, const int stride_h, const int stride_w, const int dilation_h, const int dilation_w,
                       const int deformable_group, Tensor data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  int channel_per_deformable_group = channels / deformable_group;
  // Launch
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data_im.scalar_type(), "deformable_im2col_cuda", ([&] {
    deformable_im2col_gpu_kernel<scalar_t><<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
      num_kernels, data_im.data_ptr<scalar_t>(), data_offset.data_ptr<scalar_t>(), height, width, ksize_h, ksize_w, pad_h,
      pad_w, stride_h, stride_w, dilation_h, dilation_w,
      channel_per_deformable_group, height_col, width_col, data_col.data_ptr<scalar_t>());
  }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_im2col: %s\n", hipGetErrorString(err));
    // TODO(BZ) panic
  }
}

template <typename scalar_t>
__global__ void deformable_col2im_gpu_kernel(
    const int n, const scalar_t *data_col, const scalar_t *data_offset,
    const int channels, const int height, const int width, const int kernel_h,
    const int kernel_w, const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int height_col,
    const int width_col, scalar_t *grad_im) {
  CUDA_KERNEL_LOOP(index, n) {
    const int j = (index / width_col / height_col) % kernel_w;
    const int i = (index / width_col / height_col / kernel_w) % kernel_h;
    const int c = index / width_col / height_col / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const scalar_t *data_offset_ptr = data_offset + deformable_group_index * 2 *
                                                     kernel_h * kernel_w *
                                                     height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
    const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
    const scalar_t cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const scalar_t cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const scalar_t cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              (c * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight =
              get_gradient_weight<scalar_t>(cur_inv_h_data, cur_inv_w_data, cur_h + dy,
                                  cur_w + dx, height, width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

void deformable_col2im(Tensor data_col,
                       Tensor data_offset, const int channels,
                       const int height, const int width, const int ksize_h,
                       const int ksize_w, const int pad_h, const int pad_w,
                       const int stride_h, const int stride_w,
                       const int dilation_h, const int dilation_w,
                       const int deformable_group, Tensor grad_im) {

  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * ksize_h * ksize_w * height_col * width_col;
  int channel_per_deformable_group = channels / deformable_group;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data_col.scalar_type(), "deformable_col2im_cuda", ([&] {
    deformable_col2im_gpu_kernel<scalar_t><<<GET_BLOCKS(num_kernels), CUDA_NUM_THREADS>>>(
      num_kernels, data_col.data_ptr<scalar_t>(), data_offset.data_ptr<scalar_t>(), channels, height, width, ksize_h,
      ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
      channel_per_deformable_group, height_col, width_col, grad_im.data_ptr<scalar_t>());
  }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_col2im: %s\n", hipGetErrorString(err));
    // TODO(BZ) panic
  }
}

template <typename scalar_t>
__global__ void deformable_col2im_coord_gpu_kernel(
    const int n, const scalar_t *data_col, const scalar_t *data_im,
    const scalar_t *data_offset, const int channels, const int height,
    const int width, const int kernel_h, const int kernel_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int channel_per_deformable_group, const int height_col,
    const int width_col, scalar_t *grad_offset) {
  CUDA_KERNEL_LOOP(index, n) {
    scalar_t val = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = index / width_col / height_col;
    // compute the start and end of the output

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const scalar_t *data_col_ptr = data_col + deformable_group_index *
                                               channel_per_deformable_group *
                                               width_col * height_col;
    const scalar_t *data_im_ptr =
        data_im + deformable_group_index * channel_per_deformable_group /
                      kernel_h / kernel_w * height * width;
    const scalar_t *data_offset_ptr = data_offset + deformable_group_index * 2 *
                                                     kernel_h * kernel_w *
                                                     height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos = ((col_c * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col) % kernel_w;
      int i = (col_pos / width_col / height_col / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
      const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
      scalar_t inv_h = h_in + i * dilation_h + offset_h;
      scalar_t inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h < 0 || inv_w < 0 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -1;
      }
      const scalar_t weight = get_coordinate_weight<scalar_t>(
          inv_h, inv_w, height, width, data_im_ptr + cnt * height * width,
          width, bp_dir);
      val += weight * data_col_ptr[col_pos];
      cnt += 1;
    }

    grad_offset[index] = val;
  }
}

void deformable_col2im_coord(Tensor data_col,
                             Tensor data_im, Tensor data_offset,
                             const int channels, const int height,
                             const int width, const int ksize_h,
                             const int ksize_w, const int pad_h,
                             const int pad_w, const int stride_h,
                             const int stride_w, const int dilation_h,
                             const int dilation_w, const int deformable_group,
                             Tensor grad_offset) {

  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels =
      height_col * width_col * 2 * ksize_h * ksize_w * deformable_group;
  int channel_per_deformable_group =
      channels * ksize_h * ksize_w / deformable_group;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data_col.scalar_type(), "deformable_col2im_cuda", ([&] {
    deformable_col2im_coord_gpu_kernel<scalar_t><<<GET_BLOCKS(num_kernels),CUDA_NUM_THREADS>>>(
      num_kernels, data_col.data_ptr<scalar_t>(), data_im.data_ptr<scalar_t>(), data_offset.data_ptr<scalar_t>(), channels, height, width,
      ksize_h, ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h,
      dilation_w, channel_per_deformable_group, height_col, width_col,
      grad_offset.data_ptr<scalar_t>());
  }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_col2im: %s\n", hipGetErrorString(err));
    // TODO(BZ) panic
  }
}

